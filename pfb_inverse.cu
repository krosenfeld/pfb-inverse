/*
Inverse PFB following Richard Shaw's original python/LAPACK routine: https://github.com/jrs65/pfb-inverse
Beware: This implementation runs ~4x slower than the python version on hamster!
@author Katherine Rosenfeld
@date 8/2015

To compile:
  nvcc pfb_inverse.cu -o pfb_inverse.out -lcublas -lcurand -lcufft -llapack
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hipfft/hipfft.h>


#define BENG_CHANNELS_ 16384
#define BENG_SNAPSHOTS 128
#define PI 3.14159265359

extern "C" {
 void dpbtrf_(char* uplo, int *n, int* kd, double* ab, int* ldab, int* info);
}

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d: %s\n",__FILE__,__LINE__,hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__host__ __device__ float hamming(int n, int m){
  return 0.54 - 0.46*cos(2.*PI*n/(m-1.));
}

// decimation kernel
__global__ void decimate(hipfftComplex *in, hipfftComplex *out, int M, int N){
  int tid = blockIdx.x*blockDim.x + threadIdx.x; 
  for (int i=tid; i<N; i+= gridDim.x*blockDim.x){
    if (i % M == 0) {
      out[i / M] = in[i];
    }
  }
}

// multiple kernel
__global__ void multiply(float *a, float b, int N){
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  for (int i=tid; i<N; i+=gridDim.x*blockDim.x){
    a[i] *= b;
  }
}

// cross multiply kernel
__global__ void cross_multiply(hipfftComplex *S_0x1, hipfftComplex *X0, hipfftComplex *X1, int N){
  // returns S_0x1 = X0 * conj(X1)
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  for (int i = tid; i < N; i += blockDim.x*gridDim.x){
    S_0x1[i].x = X0[i].x*X1[i].x + X0[i].y*X1[i].y;
    S_0x1[i].y = X0[i].y*X1[i].x - X0[i].x*X1[i].y;
  }
}

// compute mean along column [m x n, row major format]
__global__ void col_mean(hipfftComplex *in, int m, int n){
  int cid = blockIdx.x*blockDim.x + threadIdx.x;
  // stride along column id
  for (int i = cid; i < n; i += gridDim.x*blockDim.x){
    float avg_re = 0;
    float avg_im = 0;
    for (int j = 0 ; j < m; j++){
      avg_re += in[i + j*n].x;
      avg_im += in[i + j*n].y;
    }
      //in[i] = make_cuComplex(avg_re / m, avg_im / m);
      in[i].x = avg_re/m;
      in[i].y = avg_im/m;
  }
}


// apply window function
__global__ void window(float *in, float *out, int N){
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  for (int i=tid; i<N; i+= gridDim.x*blockDim.x){
	out[i] = in[i]*hamming(i,N);
  }
}

float corr_FXt(float *d_x0, float *d_x1, int num_samples){
  int idx,window_size = 32768*64;
  hipfftHandle plan,iplan;
  hipblasHandle_t handle;
  int batch = num_samples / window_size;
  hipfftComplex *d_S,*d_X0, *d_X1;
  dim3 blocks(64,1,1),threads(256,1,1);
  float *d_s;
  float s0x0_max, s1x1_max, corr_coeff;

  printf("%s : batch = %d \n",__FUNCTION__, batch);

  // allocate device arrays
  CUDA_CALL( hipMalloc((void **) &d_X0, (window_size/2+1)*batch*sizeof(hipfftComplex)) );
  CUDA_CALL( hipMalloc((void **) &d_X1, (window_size/2+1)*batch*sizeof(hipfftComplex)) );
  CUDA_CALL( hipMalloc((void **) &d_S,  (window_size/2+1)*batch*sizeof(hipfftComplex)) );
  CUDA_CALL( hipMalloc((void **) &d_s,  window_size*sizeof(float)) );

  // create FFT plans and cuBLAS handle
  CUFFT_CALL( hipfftPlanMany(&plan, 1, &window_size, NULL,1,0,NULL,1,0,HIPFFT_R2C,batch) );
  CUFFT_CALL( hipfftPlanMany(&iplan, 1, &window_size, NULL,1,0,NULL,1,0,HIPFFT_C2R,1) );
  CUBLAS_CALL( hipblasCreate(&handle) );

  // execute R2C FFT
  CUFFT_CALL( hipfftExecR2C(plan, d_x0, d_X0) );
  CUFFT_CALL( hipfftExecR2C(plan, d_x1, d_X1) );

  // auto-corr X0, X0
  cross_multiply<<<blocks,threads>>>(d_S,d_X0,d_X0,batch*(window_size/2+1));
  col_mean<<<blocks,threads>>>(d_S,batch,window_size/2+1);
  CUFFT_CALL( hipfftExecC2R(iplan, d_S, d_s) );
  CUBLAS_CALL( hipblasIsamax(handle, window_size, d_s, 1, &idx) );
  CUDA_CALL( hipMemcpy( &s0x0_max, d_s + (idx-1), 1*sizeof(float), hipMemcpyDeviceToHost) );

  // auto-corr X1, X1
  cross_multiply<<<blocks,threads>>>(d_S,d_X1,d_X1,batch*(window_size/2+1));
  col_mean<<<blocks,threads>>>(d_S,batch,window_size/2+1);
  CUFFT_CALL( hipfftExecC2R(iplan, d_S, d_s) );
  CUBLAS_CALL( hipblasIsamax(handle, window_size, d_s, 1, &idx) );
  CUDA_CALL( hipMemcpy( &s1x1_max, d_s + (idx-1), 1*sizeof(float), hipMemcpyDeviceToHost) );

  // cross-corr X0, X1
  cross_multiply<<<blocks,threads>>>(d_S,d_X0,d_X1,batch*(window_size/2+1));
  col_mean<<<blocks,threads>>>(d_S,batch,window_size/2+1);
  CUFFT_CALL( hipfftExecC2R(iplan, d_S, d_s) );
  CUBLAS_CALL( hipblasIsamax(handle, window_size, d_s, 1, &idx) );
  CUDA_CALL( hipMemcpy( &corr_coeff, d_s + (idx-1), 1*sizeof(float), hipMemcpyDeviceToHost) );
  printf("corr coeff: %0.4f %d \n",corr_coeff/sqrt(s1x1_max*s0x0_max), idx);


  // clean up
  CUFFT_CALL( hipfftDestroy(plan) );
  CUFFT_CALL( hipfftDestroy(iplan) );
  CUDA_CALL( hipFree(d_X0) );
  CUDA_CALL( hipFree(d_X1) );
  CUDA_CALL( hipFree(d_S) );
  CUDA_CALL( hipFree(d_s) );
  CUBLAS_CALL( hipblasDestroy(handle) );
  return corr_coeff/sqrt(s1x1_max*s0x0_max); 
}


int PPT(int nblock, int lblock, int ntap, float *d_uPPT, float *d_band_P){
// http://www.physics.orst.edu/~rubin/nacphy/lapack/routines/spbtrf.html
  double *ab, *coeff_P, *coeff_PPT;
  float  *uPPT, *band_P;
  char uplo = 'U';	// store upper triangle
  int n = nblock, kd = ntap-1, ldab = ntap, info;
  int ntsblock = nblock + ntap - 1;

  // allocate memory
  coeff_P = (double *) malloc(ntap*lblock*sizeof(double));
  coeff_PPT = (double *) malloc(lblock*ntap*sizeof(double));
  ab = (double*) malloc(ntap*nblock*sizeof(double));
  uPPT = (float*) malloc(lblock*ntap*nblock*sizeof(float));
  band_P = (float*) malloc(lblock*ntap*ntsblock*sizeof(float));

  // generate window function
  for (int i=0; i<ntap*lblock; i++){
	coeff_P[i] = hamming(i,ntap*lblock);
  }

  for (int i=0; i<lblock*ntap; i++)  coeff_PPT[i] = 0.;	// initialize array
  for (int k=0; k < ntap; k++){
    for (int j=0; j < lblock; j++){
      for (int i=0; i < ntap - k; i++){
	coeff_PPT[k*lblock + j] += coeff_P[(i+k)*lblock + j] * coeff_P[i*lblock + j];
      }
    }
  } 

  // compute Cholesky factorization of each coeff_PPT submatrix 
  // remember that lapack has column major format
  for (int i=0; i<lblock; i++){
    // band_PPT
    for (int j=0; j<ntap; j++){
      for (int k=0; k<nblock; k++){
        ab[k*ntap + j] = coeff_PPT[(ntap-1-j)*lblock + i];
      }
    }
    dpbtrf_(&uplo, &n, &kd, ab, &ldab, &info); 
    if (info != 0){
      printf("pbtrf error :%d\n",info);
    }
    for (int j=0; j<ntap; j++){
      for (int k=0; k<nblock; k++){
        uPPT[i*ntap*nblock + k*ntap + j] = (float) ab[k*ntap + j]; // cuBLAS also has column major format
      }
    }
  }

  // fill host arrays
  for (int k=0; k<lblock; k++){
    for (int j=0; j<ntap; j++){
      for (int i=0; i<ntsblock; i++){
        band_P[k*ntap*ntsblock + i*ntap + j] = (float) coeff_P[(ntap-1-j)*lblock + k];
      }
    }
  }

  // load to device
  CUDA_CALL( hipMemcpy(d_uPPT, uPPT, lblock*ntap*nblock*sizeof(float), hipMemcpyHostToDevice) );
  CUDA_CALL( hipMemcpy(d_band_P, band_P, ntsblock*lblock*ntap*sizeof(float), hipMemcpyHostToDevice) );

  free(uPPT);
  free(ab);
  free(coeff_P);
  free(coeff_PPT);
  free(band_P);
  return 1;
}

// generate pfb spectrum (doesn't actually do the polyphase bit...)
int pfb(float *d_t, int num_samples, int num_tap, int num_freq, hipfftComplex *d_s){
  int lblock = 2 * (num_freq - 1);
  int nblock = num_samples / lblock - (num_tap - 1);
  float *d_tt;
  hipfftComplex *d_ft;
  hipfftHandle plan;

  // create FFT plan
  int batch = 1;
  int fft_size = lblock*num_tap;
  CUDA_CALL( hipMalloc((void **) &d_ft, (fft_size/2+1)*sizeof(hipfftComplex)) ); 
  CUDA_CALL( hipMalloc((void **) &d_tt, fft_size*sizeof(hipfftComplex)) ); 
  CUFFT_CALL( hipfftPlanMany(&plan, 1, &fft_size,NULL,1,0,NULL,1,0,HIPFFT_R2C,batch) );

  dim3 blocks(64,1,1);
  dim3 threads(512,1,1);

  // iterate over blocks (no batches yet)
  for (int i=0; i < nblock; i++){

	// window
	window<<<blocks,threads>>>(d_t + i*lblock, d_tt, fft_size);
	CUDA_CALL(hipGetLastError());

	// execute rFFT
  	CUFFT_CALL( hipfftExecR2C(plan, d_tt, d_ft) );

	// decimate
	decimate<<<blocks,threads>>>(d_ft,d_s+i*num_freq,num_tap,fft_size/2+1);
	CUDA_CALL(hipGetLastError());
  }

  CUDA_CALL( hipFree(d_ft) );
  CUDA_CALL( hipFree(d_tt) );
  CUFFT_CALL( hipfftDestroy(plan) );
  return 1;
}

/*
 d_s is complex PFB timestream [num_snapshots, num_freqs]
*/
int inverse_pfb(hipfftComplex *d_s, int num_samples, int num_tap, int num_freq, float *d_rts){
  hipfftHandle plan;
  hipblasHandle_t handle;
  hipblasStatus_t err;
  float *d_pts, *d_foo, *d_uPPT, *d_band_P;
  hipEvent_t tic,toc;
  float elapsedTime;

  // pull out the number of blocks and their length
  int lblock = 2 * (num_freq - 1);
  int nblock = num_samples / lblock - (num_tap - 1);
  int ntsblock = nblock + num_tap - 1;
  float beta = 0.0,alpha = 1.0;

  // create CUDA events for timing
  hipEventCreate(&tic);
  hipEventCreate(&toc);

  // create cuBLAS context
  CUBLAS_CALL( hipblasCreate(&handle) );

  // generate and load coeff_P and Cholesky factorized PPT matrix to device
  CUDA_CALL( hipMalloc((void **) &d_uPPT, nblock*lblock*num_tap*sizeof(float)) );
  CUDA_CALL( hipMalloc((void **) &d_band_P, ntsblock*lblock*num_tap*sizeof(float)) );
  PPT(nblock, lblock, num_tap, d_uPPT, d_band_P);

  hipEventRecord(tic);

  // generate pseudo timestream
  CUDA_CALL( hipMalloc((void **) &d_pts, nblock*lblock*sizeof(float)) );
  CUDA_CALL( hipMalloc((void **) &d_foo, ntsblock*lblock*sizeof(float)) );
  CUFFT_CALL( hipfftPlanMany(&plan, 1, &lblock, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2R, nblock) );
  CUFFT_CALL( hipfftExecC2R(plan, d_s, d_foo) );

  // calculate correlation using pseudo timestream
  float corr_coeff = corr_FXt(d_rts,d_foo, num_samples);

  // transpose the nblock x lblock spectrum to lblock x nblock
  // cufft assumes row major jormat, cublas assumes collumn major format
  // http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-geam
  err = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			nblock, lblock,
			&alpha, d_foo, lblock,
			&beta, NULL, nblock,
			d_pts, nblock);
  if (err != HIPBLAS_STATUS_SUCCESS){
    printf("Error at %s:%s:%d\n",__FILE__,__FUNCTION__,__LINE__);
  }

  // multiple pseudo-timestream by 1./lblock (to rescale inverse FFT)
  dim3 blocks(64,1,1);
  dim3 threads(512,1,1);
  multiply<<<blocks,threads>>>(d_pts,1./lblock,lblock*nblock);

  // probably want to batch this or use streams
  for (int i = 0; i < lblock; i++){

    // solve for intermediate vector
    // http://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-tbsv
    err = hipblasStbsv(handle,HIPBLAS_FILL_MODE_UPPER,
		HIPBLAS_OP_T,HIPBLAS_DIAG_NON_UNIT,
		nblock,num_tap-1,
		d_uPPT+i*nblock*num_tap,num_tap,
		d_pts+i*nblock,1);
    if (err != HIPBLAS_STATUS_SUCCESS){
      printf("Error at %s:%s:%d\n",__FILE__,__FUNCTION__,__LINE__);
    }

    err = hipblasStbsv(handle,HIPBLAS_FILL_MODE_UPPER,
		HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,
		nblock,num_tap-1,
		d_uPPT+i*nblock*num_tap,num_tap,
		d_pts+i*nblock,1);
    if (err != HIPBLAS_STATUS_SUCCESS){
      printf("Error at %s:%s:%d\n",__FILE__,__FUNCTION__,__LINE__);
    }

    // project back onto time-stream
    // http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gbmv
    err = hipblasSgbmv(handle,HIPBLAS_OP_T,
		nblock, ntsblock, 0, num_tap-1,
		&alpha,d_band_P+i*num_tap*ntsblock,num_tap,
		d_pts+i*nblock, 1, 
		&beta, d_foo+i*ntsblock, 1
	);
		//&beta, d_rts+i*ntsblock, 1
    if (err != HIPBLAS_STATUS_SUCCESS){
      printf("Error at %s:%s:%d\n",__FILE__,__FUNCTION__,__LINE__);
    }

  }

  // now transpose lblock x ntsblock to ntsblock x lblock
  // but remember that cublas is column major 
  err = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			lblock, ntsblock,
			&alpha, d_foo, ntsblock,
			&beta, NULL, lblock,
			d_rts, lblock);
  if (err != HIPBLAS_STATUS_SUCCESS){
    printf("Error at %s:%s:%d\n",__FILE__,__FUNCTION__,__LINE__);
  }
		 

  hipEventRecord(toc);
  hipEventSynchronize(toc);
  hipEventElapsedTime(&elapsedTime,tic,toc);
  printf("inverse-pfb (gpu only): %f\n",elapsedTime);

  CUDA_CALL( hipEventDestroy(tic) );
  CUDA_CALL( hipEventDestroy(toc) );
  CUDA_CALL( hipFree(d_pts) );
  CUDA_CALL( hipFree(d_foo) );
  CUDA_CALL( hipFree(d_uPPT) );
  CUDA_CALL( hipFree(d_band_P) );
  CUBLAS_CALL( hipblasDestroy(handle) );
  return 1;
}


int main(int argc, char* argv[]){
  int num_beng_frames = 2;
  int num_tap = 4, num_freq = BENG_CHANNELS_ + 1;
  float elapsedTime;
  float *d_ts, *d_rts, *ts, *rts;
  hipfftComplex *d_s;
  hipEvent_t tic, toc;
  hiprandGenerator_t gen;

  int num_samples = 2*BENG_CHANNELS_*(BENG_SNAPSHOTS*num_beng_frames + num_tap - 1);
  int lblock = 2 * (num_freq - 1);
  int nblock = num_samples / lblock - (num_tap - 1);

  printf("num_samples=%d\n",num_samples);
  printf("num_freqs=%d\n",num_freq);
  printf("lblock=%d\n",lblock);
  printf("nblock=%d\n",nblock);

  // create events
  CUDA_CALL( hipEventCreate(&tic) );
  CUDA_CALL( hipEventCreate(&toc) );

  // allocate device memory
  CUDA_CALL( hipMalloc((void **) &d_ts, num_samples*sizeof(float)) );
  CUDA_CALL( hipMalloc((void **) &d_s, nblock*num_freq*sizeof(hipfftComplex)) ); 

  // generate data
  CURAND_CALL( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  CUDA_CALL( hipEventRecord(tic) );
  CURAND_CALL(hiprandGenerateNormal(gen, d_ts, num_samples, 0., 1.) );
  CUDA_CALL( hipEventRecord(toc) );
  CUDA_CALL( hipEventSynchronize(toc) );
  CUDA_CALL( hipEventElapsedTime(&elapsedTime,tic,toc) ); 
  fprintf(stdout, "generating %d random numbers took %f ms\n",num_samples,elapsedTime);

  // pfb
  CUDA_CALL( hipEventRecord(tic) );
  pfb(d_ts, num_samples, num_tap, num_freq, d_s);
  CUDA_CALL( hipEventRecord(toc) );
  CUDA_CALL( hipEventSynchronize(toc) );
  CUDA_CALL( hipEventElapsedTime(&elapsedTime,tic,toc) ); 
  fprintf(stdout, "pfb took %f ms\n",elapsedTime);

  // inverse pfb
  CUDA_CALL( hipMalloc((void **) &d_rts, num_samples*sizeof(float)) );
  CUDA_CALL( hipEventRecord(tic) );
  inverse_pfb(d_s, num_samples, num_tap, num_freq, d_rts);
  CUDA_CALL( hipEventRecord(toc) );
  CUDA_CALL( hipEventSynchronize(toc) );
  CUDA_CALL( hipEventElapsedTime(&elapsedTime,tic,toc) ); 
  fprintf(stdout, "inverse-pfb took %f ms\n",elapsedTime);

  // compute the correlation coefficient here:
  CUDA_CALL( hipEventRecord(tic) );
  float corr_coeff = corr_FXt(d_rts,d_ts, num_samples);
  CUDA_CALL( hipEventRecord(toc) );
  CUDA_CALL( hipEventSynchronize(toc) );
  CUDA_CALL( hipEventElapsedTime(&elapsedTime,tic,toc) ); 
  fprintf(stdout, "FXcorr took %f ms\n",elapsedTime);

#if 0
  // write time streams to file
  ts =  (float*) malloc(num_samples*sizeof(float));
  rts = (float*) malloc(num_samples*sizeof(float)); 
  CUDA_CALL( cudaMemcpy(ts, d_ts, num_samples*sizeof(float), cudaMemcpyDeviceToHost) );
  CUDA_CALL( cudaMemcpy(rts, d_rts, num_samples*sizeof(float), cudaMemcpyDeviceToHost) );

  FILE *pFile;
  pFile = fopen("ts.txt","w");
  for (int i=0; i < num_samples; i++){
    fprintf(pFile,"%e %e\n",ts[i], rts[i]);
  }
  fclose(pFile);

  free(ts);
  free(rts);
#endif

  // clean up
  CURAND_CALL( hiprandDestroyGenerator(gen) );
  CUDA_CALL( hipEventDestroy(tic) );
  CUDA_CALL( hipEventDestroy(toc) );
  CUDA_CALL( hipFree(d_ts) );
  CUDA_CALL( hipFree(d_s) );
  CUDA_CALL( hipFree(d_rts) );
  fprintf(stdout,"done!\n");
}
